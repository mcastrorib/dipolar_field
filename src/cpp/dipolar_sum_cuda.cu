#include "hip/hip_runtime.h"
#include "dipolar_sum_cuda.h"

// Kernels
__global__ void m_kernel(void)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	printf("Hello from thread %d \n", idx);
}

// Wrappers
void dipolar_sum_analysis_periodic_cuda()
{
	printf("periodic analysis\n");
	m_kernel<<<32,1>>>();
	hipDeviceSynchronize();
	hipDeviceReset();
}

void dipolar_sum_analysis_volume_cuda()
{
	printf("volume analysis");
	m_kernel<<<32,1>>>();
	hipDeviceSynchronize();
	hipDeviceReset();
}